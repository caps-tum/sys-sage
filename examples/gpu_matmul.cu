#include <hip/hip_runtime.h>
#include <iostream>


#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

void dstMatrix(float *A, int n);
void zeroMatrix(float *A, int n);
void printMatrix(const char* name, float *A, int n);
void checkMatrix(float *to_check, float *reference, int n);
void checkCUDAError(const char *msg);
__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n, int tile_size);
__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n, int tile_size, int core_tile_size);

void usage(char* argv0)
{
    std::cerr << "usage: " << argv0 << " <gpu-topo path>" << std::endl;
    std::cerr << "       or" << std::endl;
    std::cerr << "       " << argv0 << " (uses predefined paths which may be incorrect.)" << std::endl;
    return;
}


__global__ void matrixMultKernel_cache2(float* Ad, float* Bd, float* Cd, int n, int tile_size_y, int tiles_per_warp)
{
    // __shared__ float Ad_shared[36][160];
    // __shared__ float Bd_shared[36][160];

    // extern __shared__ char array[];
    // float *Ad_shared = (float *)array;
    // float *Bd_shared = blockDim.x * blockDim.y * sizeof(float) + array;
    //
    // for(int tile_col_num = blockidx.x ; tile_col_num < num_tiles_col; tile_col_num += oversubscribe)
    // {
    //     int tileJ=tile_col_num;
    //     int tileI=blockIdx.y;
    //
    //     int elemJ=threadIdx.x;//cols
    //     int elemI=threadIdx.y;//rows
    //
    //     float resCell = 0;
    //
    //     for (int t=0; t<num_tiles_col; t++)
    //     {
    //       //j*n = row j + i=column i; A for each column block; B for each row block
    //       //load the blocks to the memory
    //       Ad_shared[elemI*tile_size_cols + elemJ] = Ad[(tileI*tile_size_rows+elemI)*n + elemJ + t*tile_size_cols];
    //       Bd_shared[elemI*tile_size_cols + elemJ] = Bd[tileJ*tile_size_cols + elemJ + elemI*n + t*n*tile_size_rows];
    //       __syncthreads();
    //
    //       for(int k=0; k<tile_size_cols; k++)
    //       {
    //          resCell += Ad_shared[elemI*tile_size_cols + k]*Bd_shared[k*tile_size_cols + elemJ];
    //       }
    //       __syncthreads();
    //     }
    //     Cd[(tileI*tile_size_rows+elemI)*n + tileJ*tile_size_cols+elemJ]=resCell;
    // }
}

__global__ void matrixMultKernel_cache(float* Ad, float* Bd, float* Cd, int n, int tile_size_rows, int tile_size_cols, int split_size_rows, int splits_per_tile_cols, int num_tiles_col )
{
    extern __shared__ float array[];
    float *Ad_shared = array;
    float *Bd_shared = &array[tile_size_rows * tile_size_cols];

    int thread_in_warp = threadIdx.x;
    int warp_id = threadIdx.y;

    float* resArray = new float(split_size_rows * splits_per_tile_cols);

    // if(warp_id == 0 && thread_in_warp == 0)
    // {
    //     printf("starting block %d\n", blockIdx.x);
    // }
    if(blockIdx.x != 0)
    {
        clock_t start_clock = clock();
        clock_t clock_offset = 0;
        while (clock_offset < 100000000)
        {
            clock_offset = clock() - start_clock;
        }
    }
// else
// {
    if(blockIdx.x * tile_size_rows >= n)
    {
        // if(warp_id == 0 && thread_in_warp == 0)
        //     printf("block %d skipping -- indexes %d and on\n", blockIdx.x, blockIdx.x * tile_size_rows);
        return;
    }

    if(blockIdx.x == 0 && warp_id == 0 && thread_in_warp == 0)
    {
        for(int row = 0; row < n; row++ )
        {
            for(int col = 0; col < n ; col++)
            {
                printf("%d-%d  ", Ad[row*n + col], Bd[row*n + col]);
            }
            printf("\n");
        }
    }

    for(int tile_col_num = 0 ; tile_col_num < num_tiles_col; tile_col_num++)
    {//extra loop to iterate over tiles in a col

        int tile_col=tile_col_num;//J
        int tile_row=blockIdx.x;//I

        if(warp_id == 0 && thread_in_warp == 0)
            printf("block %d --(%d) processing tile col indexes %d to %d\n", blockIdx.x, tile_size_rows * tile_size_cols, tile_col_num * tile_size_cols, (tile_col_num+1) * tile_size_cols -1);

        for(int i = 0; i < split_size_rows * splits_per_tile_cols; i++ )
        {
            resArray[i] = 0;
        }

        if(warp_id == 0 && thread_in_warp == 0)
            printf("block %d --(%d) processing tile col indexes %d to %d\n", blockIdx.x, tile_size_rows * tile_size_cols, tile_col_num * tile_size_cols, (tile_col_num+1) * tile_size_cols -1);

        for (int t=0; t<num_tiles_col; t++)
        {
            for(int row = 0; row < split_size_rows; row++ )
            {
                int elem_row = warp_id * split_size_rows + row;
                if(tile_row*tile_size_rows+elem_row >= n) break;
                for(int split_col = 0; split_col < splits_per_tile_cols ; split_col++)
                {
                    int elem_col = split_col*WARP_SIZE + thread_in_warp;
                    if(tile_col*tile_size_cols+elem_col >= n) break;
                    Ad_shared[elem_row*tile_size_cols + elem_col] = Ad[(tile_row * tile_size_rows + elem_row)*n + elem_col + t * tile_size_cols];
                    Bd_shared[elem_row*tile_size_cols + elem_col] = Bd[ tile_col*tile_size_cols + elem_col  + elem_row * n + t * n * tile_size_rows];
                }
            }
            __syncthreads();



            if(warp_id == 0 && thread_in_warp == 0)
                printf("block %d tile %d -- filled shared array t %d\n", blockIdx.x, tile_col_num, t);

            if(warp_id == 0 && thread_in_warp == 0)
            {
                for(int row = 0; row < tile_size_rows; row++ )
                {
                    for(int col = 0; col < 12 ; col++)
                    {
                        printf("%d ", Ad_shared[row*tile_size_cols + col]);
                    }
                    printf("\n");
                }
            }

            // if(warp_id == 0 && thread_in_warp == 0)
            // {
            //     for(int row = 0; row < split_size_rows*4; row++ )
            //     {
            //         for(int split_col = 0; split_col < splits_per_tile_cols*tile_size_cols; split_col++)
            //         {
            //             printf("%d-%d  ", Ad_shared[row*splits_per_tile_cols*tile_size_cols + split_col], Bd_shared[row*splits_per_tile_cols*tile_size_cols + split_col]);
            //         }
            //         printf("\n");
            //     }
            // }
            for(int k=0; k<tile_size_cols; k++)
            {
                // if(warp_id == 0 && thread_in_warp == 0)
                //     printf("block %d tile %d --k=%d \n", blockIdx.x, tile_col_num, k);
                for(int row = 0; row < split_size_rows; row++ )
                {
                    int elem_row = warp_id * split_size_rows + row;
                    // if(thread_in_warp == 0)
                    //     printf("block %d tile %d --k=%d -- elem_row=%d \n", blockIdx.x, tile_col_num, k, elem_row);
                    if(tile_row*tile_size_rows+elem_row >= n) break;
                    // for(int col = 0; col < tile_size_cols; col++)
                    // {
                    //     if(warp_id == 0 && thread_in_warp == 0)
                    //     {
                    //         if(tile_col*tile_size_cols+col >= n) break;
                    //         resArray[row*splits_per_tile_cols + split_col] += Ad_shared[elem_row * tile_size_cols + k] * Bd_shared[k * tile_size_rows + elem_row];
                    //     }
                    // }

                    // for(int split_col = 0; split_col < splits_per_tile_cols; split_col++)
                    // {
                    //     int elem_col = split_col*WARP_SIZE + thread_in_warp;
                    //     // if(warp_id == 0 && thread_in_warp == 31)
                    //     //     printf("block %d tile %d --k=%d -- elem_row=%d  -- elem_col=%d \n", blockIdx.x, tile_col_num, k, elem_row, elem_col);
                    //     if(tile_col*tile_size_cols+elem_col >= n) break;
                    //     if(elem_row * tile_size_cols + k >= tile_size_rows * tile_size_cols)
                    //         printf("!!!!! block %d tile %d (%d)--elem_row %d * tile_size_cols %d + k %d \n", blockIdx.x, tile_col_num, tile_size_rows * tile_size_cols, elem_row, tile_size_cols, k);
                    //     if(k * tile_size_rows + elem_row >= tile_size_rows * tile_size_cols)
                    //         printf("!!!!2 block %d tile %d (%d)--k %d * tile_size_rows %d + elem_row %d \n", blockIdx.x, tile_col_num, tile_size_rows * tile_size_cols, k, tile_size_rows, elem_row);
                    //
                    //     resArray[row*splits_per_tile_cols + split_col] += Ad_shared[elem_row * tile_size_cols + k] * Bd_shared[k * tile_size_rows + elem_row];
                    // }
                }
            }
            if(warp_id == 0 && thread_in_warp == 0)
                printf("block %d tile %d -- finished computation number t %d\n", blockIdx.x, tile_col_num, t);
            __syncthreads();

        }
        //fan-in from all threads in one warp that have co-computed the same indexes (one row)
        for(int i = 0; i< split_size_rows * splits_per_tile_cols; i++)
        {
            for (int offset = 16; offset > 0; offset /= 2)
            {
                resArray[i] += __shfl_down_sync(FULL_MASK, resArray[i], offset);
            }
        }

        if(thread_in_warp == 0)
        {
            for(int row = 0; row < split_size_rows; row++ )
            {
                int elem_row = warp_id * split_size_rows + row;
                if(tile_row*tile_size_rows+elem_row >= n) break;
                for(int split_col = 0; split_col < splits_per_tile_cols; split_col++)
                {
                    int elem_col = split_col*WARP_SIZE + thread_in_warp;
                    if(tile_col*tile_size_cols+elem_col >= n) break;
                    Cd[(tile_row * tile_size_rows +elem_row) * n + tile_col * tile_size_cols + elem_col] = resArray[row * splits_per_tile_cols + split_col ];
                }
            }
            if(warp_id == 0 && thread_in_warp == 0)
                printf("block %d finishing tile %d \n", blockIdx.x,tile_col_num);
        }
    }
// }
}

int main(int argc, const char *argv[]) {

    // //create root Topology and one node
    // Topology* topo = new Topology();
    // Node* n = new Node(topo,1);
    //
    // cout << "-- Parsing gpu-topo benchmark from file " << gpuTopoPath << endl;
    // if(parseGpuTopo((Component*)n, gpuTopoPath, 0, ";") != 0) { //adds topo to a next node
    //     return 1;
    // }
    // cout << "-- End parseGpuTopo" << endl;

    float *A,*B,*C,*D; /* arrays for matrices */
    //int n; /* n=matrix size */
    int n = atoi(argv[1]);
    printf("Matrix mult. of size %d : \n", n);

    int size = n*n*sizeof(float);
    A = (float *) malloc(size);
    dstMatrix(A,n);
    B = (float *) malloc(size);
    dstMatrix(B,n);
    C = (float *) malloc(size);
    zeroMatrix(C,n);
    D = (float *) malloc(size);
    zeroMatrix(D,n);

    float *Ad, *Bd, *Cd, *Dd;

    hipMalloc((void**)&Ad, size); checkCUDAError("allocate memory for A");
    hipMalloc((void**)&Bd, size); checkCUDAError("allocate memory for B");
    hipMalloc((void**)&Cd, size); checkCUDAError("allocate memory for C");
    hipMalloc((void**)&Dd, size); checkCUDAError("allocate memory for D");

    hipMemcpy(Ad,A, size, hipMemcpyHostToDevice); checkCUDAError("memory of A not transferred");
    hipMemcpy(Bd,B, size, hipMemcpyHostToDevice); checkCUDAError("memory of B not transferred");
    hipMemcpy(Cd,C, size, hipMemcpyHostToDevice); checkCUDAError("memory of C not transferred");
    hipMemcpy(Dd,D, size, hipMemcpyHostToDevice); checkCUDAError("memory of D not transferred");

    //n=4000
    int gpu_num_cores = 3840;
    int gpu_num_sm = 30;
    int oversubscribe = 2;
    int shared_mem_size = 96 * 1024 * 0.9;

    int num_cores_per_sm = gpu_num_cores / gpu_num_sm;
    int num_warps_per_sm = num_cores_per_sm / WARP_SIZE;
    printf("matrix size %d * %d , num warps %d\n", n, n, num_warps_per_sm*gpu_num_sm);

    int num_row_splits = num_warps_per_sm * gpu_num_sm * oversubscribe;
    int split_size_rows = (n +num_row_splits-1)/num_row_splits;
    int split_size_cols = WARP_SIZE;
    printf("split dims %d rows x %d cols \n", split_size_rows, split_size_cols);

    int splits_per_tile_rows = num_warps_per_sm;
    int tile_size_rows = split_size_rows * num_warps_per_sm;

    int splits_per_tile_cols = shared_mem_size / (2 * oversubscribe * tile_size_rows * WARP_SIZE * sizeof(float));
    int tile_size_cols = splits_per_tile_cols * WARP_SIZE;

    printf("num splits per tile = %d row x %d col \n", splits_per_tile_rows, splits_per_tile_cols);
    printf("\ntile dim %d row x %d col = %d bytes \n",tile_size_rows, tile_size_cols, tile_size_cols * tile_size_rows * 4);
    printf("grid: %d x %d-row split-sets \n", gpu_num_sm * oversubscribe, split_size_rows*num_warps_per_sm);
    int num_tiles_col = (n+tile_size_cols-1) / tile_size_cols;
    int num_tiles_row = (n+tile_size_rows-1) / tile_size_rows;
    dim3 block_dim(WARP_SIZE, num_warps_per_sm);
    dim3 grid_dim(oversubscribe * gpu_num_sm);//gpu_num_sm * oversubscribe
    //dim3 grid_dim(num_tiles_col, num_tiles_row);//gpu_num_sm * oversubscribe
    unsigned block_shared_mem_size = tile_size_rows * tile_size_cols * sizeof(float) * 2;
    matrixMultKernel_cache<<<grid_dim,block_dim, block_shared_mem_size>>>(Ad,Bd,Cd,n,tile_size_rows, tile_size_cols, split_size_rows, splits_per_tile_cols, num_tiles_col);
    checkCUDAError("matrixMultKernel_cache failed");
    hipMemcpy(C,Cd, size, hipMemcpyDeviceToHost); checkCUDAError("memory of D not transferred back");

    printMatrix("C", C, n);
    //checkMatrix(D,C,n);
//////////
    //
    // int tile_size_rows = WARP_SIZE;//TODO threads_per_sm?
    // int tile_size_cols = shared_mem_size / (2 * oversubscribe * tile_size_rows * sizeof(float));
    // printf("\ntile dim %d x %d = %d bytes \n",tile_size_rows, tile_size_cols, tile_size_cols * tile_size_rows * 4);
    //
    // dim3 block_dim(tile_size_cols, tile_size_rows);
    // dim3 grid_dim(oversubscribe, gpu_num_sm * num_warps_per_sm);//gpu_num_sm * oversubscribe
    // unsigned shared_mem_size = tile_size_rows * tile_size_cols * sizeof(float) * 2;
    // matrixMultKernel_cache2<<<dimGrid,dimBlock, shared_mem_size >>>(Ad,Bd,Cd,n);
    //

    return 0;


    //
    //
    // int num_cores = 2000;
    // int tile_size_bytes = 24588/64;// = //24588/64 B per core
    //
    // float *A,*B,*C,*D; /* arrays for matrices */
    // //int n; /* n=matrix size */
    // n = atoi(argv[1]);
    // printf("Matrix mult. of size %d : \n", n);
    //
    // int size = n*n*sizeof(float);
    // A = (float *) malloc(size);
    // dstMatrix(A,n);
    // B = (float *) malloc(size);
    // dstMatrix(B,n);
    // C = (float *) malloc(size);
    // zeroMatrix(C,n);
    // D = (float *) malloc(size);
    // zeroMatrix(D,n);
    //
    // float *Ad, *Bd, *Cd, *Dd;
    //
    // cudaMalloc((void**)&Ad, size); checkCUDAError("allocate memory for A");
    // cudaMalloc((void**)&Bd, size); checkCUDAError("allocate memory for B");
    // cudaMalloc((void**)&Cd, size); checkCUDAError("allocate memory for C");
    // cudaMalloc((void**)&Dd, size); checkCUDAError("allocate memory for D");
    //
    // cudaMemcpy(Ad,A, size, cudaMemcpyHostToDevice); checkCUDAError("memory of A not transferred");
    // cudaMemcpy(Bd,B, size, cudaMemcpyHostToDevice); checkCUDAError("memory of B not transferred");
    // cudaMemcpy(Cd,C, size, cudaMemcpyHostToDevice); checkCUDAError("memory of C not transferred");
    // cudaMemcpy(Dd,D, size, cudaMemcpyHostToDevice); checkCUDAError("memory of D not transferred");
    //
    // int num_cores_1dim = sqrt(num_cores);
    // if(num_cores_1dim * num_cores_1dim < num_cores)
    //     num_cores_1dim+=1;
    //
    // int tile_size = n/num_cores_1dim;
    // if(tile_size * num_cores_1dim < n)
    //     tile_size+=1;
    //
    //
    // dim3 dimBlock((num_cores_1dim>32?32:num_cores_1dim), (num_cores_1dim>32?32:num_cores_1dim));
    // dim3 dimGrid( (num_cores_1dim%32==0?num_cores_1dim/32:num_cores_1dim/32+1), (num_cores_1dim%32==0?num_cores_1dim/32:num_cores_1dim/32+1) );
    // printf("launching kernel: matrix %d x %d, each thread computes tile=%d x %d, there is %d cores in each dimension (blockDim %d, gridDim %d) \n", n, n, tile_size, tile_size, num_cores_1dim, (num_cores_1dim>32?32:num_cores_1dim), (num_cores_1dim%32==0?num_cores_1dim/32:num_cores_1dim/32+1) );
    //
    // matrixMultKernel_global<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n, tile_size);
    // cudaMemcpy(C,Cd, size, cudaMemcpyDeviceToHost); checkCUDAError("memory of C not transferred back");
    //
    // printf("matrixMultKernel_global done \n");
    //
    // matrixMultKernel_tiled<<<dimGrid,dimBlock>>>(Ad,Bd,Dd,n, tile_size, 7);
    // cudaMemcpy(D,Dd, size, cudaMemcpyDeviceToHost); checkCUDAError("memory of D not transferred back");
    //
    // //printMatrix("C", C, n);
    // checkMatrix(D,C,n);
    //
    // cudaFree(Ad);
    // cudaFree(Bd);
    // cudaFree(Cd);
    //
    // return(0);
    //


}

__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n, int tile_size)
{
    int first_row_index = threadIdx.x * tile_size * n;
    int first_col_index = threadIdx.y * tile_size;

//    if(threadIdx.x==19)
//        printf("index first_row 7, accessing elems row %d to %d col %d to %d \n", first_row_index/n, first_row_index/n +tile_size, first_col_index, first_col_index + tile_size);

    for(int row=0; row < tile_size && row + threadIdx.x * tile_size < n ; row++)
    {
        for(int col=0; col < tile_size && col + threadIdx.y * tile_size < n; col++)
        {
            for(int j=0; j < n ; j++)
            {
                Cd[first_row_index + row*n + first_col_index + col] += Ad[first_row_index + row*n + j]* Bd[j*n + first_col_index + col];
            }
        }
    }
}


__global__ void matrixMultKernel_optimized(float* Ad, float* Bd, float* Cd, int n, int tile_size_y, int tiles_per_warp)
{
    // cuda_warps = 8*GPU_cores / 32
    // cuda_warps/SM = cores-per-SM/32*8
    // 1 tile - x=to fit into shared memory on SM, y=32
    // num-tiles = n/x * n/y
    // num-tiles-per-warp = num-tiles/cuda_warps

    // int row_in_tile = threadIdx.x;
    //
    // __shared__ float Ads[32][tile_size_y];
    // __shared__ float Bds[32][tile_size_y];
    //
    // float c_part = 0;
    // for(int col = 0; col < tile_size_y; col ++)
    // {
    //     c_part += Ad[ row_in_tile * n + col ] * B[ row_in_tile * n + col ]
    // }
}


__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n, int tile_size, int core_tile_size)
{
    int first_row_index = threadIdx.x * tile_size * n;
    int first_col_index = threadIdx.y * tile_size;

    for(int row=0; row < tile_size && row + threadIdx.x * tile_size < n ; row+=core_tile_size)
    {
        for(int col=0; col < tile_size && col + threadIdx.y * tile_size < n; col+=core_tile_size)
        {
            for(int j=0; j < n ; j+=core_tile_size)
            {
                for(int row_tile=row; row_tile < row+core_tile_size && row_tile + threadIdx.x * tile_size < n ; row_tile++)
                {
                    for(int col_tile=col; col_tile < col+core_tile_size && col_tile + threadIdx.y * tile_size < n; col_tile++)
                    {
                        for(int j_tile=j; j_tile < j+core_tile_size && j_tile < n ; j_tile++)
                        {
                            Cd[first_row_index + row_tile*n + first_col_index + col_tile] += Ad[first_row_index + row_tile*n + j_tile]* Bd[j_tile*n + first_col_index + col_tile];
                        }
                    }
                }
            }
        }
    }
}


/*
 * helper function to check for errors in CUDA calls
 * source: NVIDIA
 */
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();

    if( hipSuccess != err) {
        fprintf(stderr, "\nCuda error (%s): %s.\n", msg, hipGetErrorString( err) );
        //exit(-1);
    }
}

/* print Matrix */
void printMatrix(const char* name, float *A, int n)
{
   int i,k;

   printf("Matrix %s (size %d)\n",name,n);

   for (i=0; i<n; i++) {
     for (k=0; k<n; k++) {
       printf("%f ", A[i*n+k]);
     }

     printf("\n");
   }
}

void checkMatrix(float *to_check, float *reference, int n)
{
    int i,k;
    for (i=0; i<n; i++) {
      for (k=0; k<n; k++) {
        if(to_check[i*n+k] != reference[i*n+k]){
            printf("[%d,%d] %f vs ref %f \n", i,k,to_check[i*n+k],  reference[i*n+k] );
            return;
        }
      }
  }
}

/* set Matrix values to zero */
void zeroMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
	    A[i*n+k] = 0;
}

/* initialise Matrix: discrete Sine Transform */
void dstMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
        A[i*n+k] = i+k+1;
	    //A[i*n+k] = sin( ((i+1)*(k+1)*M_PI)/(n+1));
}
