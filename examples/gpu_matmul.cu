#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

void dstMatrix(float *A, int n);
void zeroMatrix(float *A, int n);
void printMatrix(const char* name, float *A, int n);
void checkMatrix(float *to_check, float *reference, int n);
void checkCUDAError(const char *msg);
__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n, int tile_size);
__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n, int tile_size, int core_tile_size);
void CPU_matrixMult(float *A, float *B, float *C, int n, int repeats);

void usage(char* argv0)
{
    std::cerr << "usage: " << argv0 << " <gpu-topo path>" << std::endl;
    std::cerr << "       or" << std::endl;
    std::cerr << "       " << argv0 << " (uses predefined paths which may be incorrect.)" << std::endl;
    return;
}
using namespace std::chrono;

__global__ void matrixMultKernel_cache(float* Ad, float* Bd, float* Cd, int n, int tile_size_rows, int tile_size_cols, int split_size_rows, int splits_per_tile_cols, int num_tiles_col, int* spinlock )
{
    extern __shared__ float array[];
    float *Ad_shared = array;
    float *Bd_shared = &array[tile_size_rows * tile_size_cols];

    int thread_in_warp = threadIdx.x;
    int warp_id = threadIdx.y;

    float* resArray = (float*)malloc(sizeof(float)*split_size_rows * tile_size_rows);

    if(blockIdx.x * tile_size_rows >= n){
         // if(warp_id == 0 && thread_in_warp == 0)
         //     printf("!!block %d skipping -- indexes %d and on\n", blockIdx.x, blockIdx.x * tile_size_rows);
        return;
    }

    // while (* spinlock < blockIdx.x*10) {
    // }

    for(int tile_offset_col = 0 ; tile_offset_col < n; tile_offset_col+=tile_size_rows)
    {//one loop of this fills tile_size_rows*tile_size_rows matrix

        int tile_offset_row=blockIdx.x*tile_size_rows;//I

        for(int i = 0; i < split_size_rows * tile_size_rows; i++ )
        {
            resArray[i] = 0;
        }

        int a_num_rows = ((tile_offset_row+tile_size_rows <= n) ? tile_size_rows : n - tile_offset_row);
        int b_num_cols = ((tile_offset_col + tile_size_rows <= n) ? tile_size_rows : n - tile_offset_col);

        for (int t=0; t<num_tiles_col; t++)
        {
            int a_num_cols_b_num_rows = ((tile_size_cols*(t+1) <= n) ? tile_size_cols : n - tile_size_cols*t );

            // if(warp_id == 0 && thread_in_warp == 0)
            //     printf("2 block %d tile col %d, t %d -- a=%dx%d , b=%dx%d -- A[%d-%d, %d-%d], B[%d-%d, %d-%d]\n", blockIdx.x, tile_offset_col, t, a_num_rows, a_num_cols_b_num_rows, a_num_cols_b_num_rows, b_num_cols,      tile_offset_row, tile_offset_row+a_num_rows, t*tile_size_cols, t*tile_size_cols+a_num_cols_b_num_rows,        t*tile_size_cols, t*tile_size_cols + a_num_cols_b_num_rows,tile_offset_col,tile_offset_col+b_num_cols);

            for(int split_col = 0; split_col < splits_per_tile_cols ; split_col++)
            {
                int elem_col = split_col*WARP_SIZE + thread_in_warp;
                if(elem_col >= a_num_cols_b_num_rows) break;
                for(int row = 0; row < split_size_rows; row++ )
                {
                    int elem_row = warp_id * split_size_rows + row;
                    if(elem_row < a_num_rows)
                        Ad_shared[elem_row*tile_size_cols + elem_col] = Ad[(tile_offset_row + elem_row)*n + t*tile_size_cols + elem_col];
                    if(elem_row < b_num_cols)
                        Bd_shared[elem_row*tile_size_cols + elem_col] = Bd[elem_row + tile_offset_col + (t*tile_size_cols + elem_col)*n];
                }
            }
            __syncthreads();


            for(int k=0; k < a_num_rows - split_size_rows*warp_id && k< split_size_rows  ; k++)
            {
                for(int l = 0; l<b_num_cols ; l++)
                {
                    for(int m = thread_in_warp; m<a_num_cols_b_num_rows; m+=WARP_SIZE)
                    {
                        //printf("block %d thread %d tile col %d --temp_res[%d][%d][%d](%f)+=As[%d][%d][%d](%f) * Bs[%d][%d][%d](%f) \n", blockIdx.x,  warp_id*WARP_SIZE+thread_in_warp, tile_offset_col, k,l,k*tile_size_rows + l,resArray[k*tile_size_rows + l], split_size_rows*warp_id + k,m,(split_size_rows*warp_id + k)*tile_size_cols + m,Ad_shared[(split_size_rows*warp_id + k)*tile_size_cols + m], l,m, l * tile_size_cols + m, Bd_shared[l * tile_size_cols + m]);
                        resArray[k*tile_size_rows + l] += Ad_shared[(split_size_rows*warp_id + k) * tile_size_cols + m] * Bd_shared[l * tile_size_cols + m];
                        //resArray[k*tile_size_rows + l] += Ad_shared[k*tile_size_cols + m] * Bd_shared[l * tile_size_cols + m];
                    }
                }
            }
            __syncthreads();

        }

        //fan-in from all threads in one warp that have co-computed the same indexes (one row)
        for(int i = 0; i < split_size_rows * tile_size_rows; i++ )
        {
            for (int offset = 16; offset > 0; offset /= 2)
            {
                resArray[i] += __shfl_down_sync(FULL_MASK, resArray[i], offset,32);
            }
        }
        __syncthreads();
        // for(int row = 0; row < tile_size_rows; row++ )
        // {
        //     for(int col = 0; col < split_size_rows ; col++)
        //     {
        //         if(resArray[row*split_size_rows + col] != 0 && thread_in_warp==0)
        //             printf("thread %d [%d,%d] _%.0f \n", warp_id*WARP_SIZE+thread_in_warp, row, col, resArray[row*split_size_rows + col]);
        //     }
        // }

        if(thread_in_warp == 0)
        {
            for(int k=0; k < a_num_rows - split_size_rows*warp_id && k< split_size_rows  ; k++)
            {
                for(int l = 0; l<b_num_cols ; l++)
                {
                     //printf("6 -----===== block %d thread %d filling index [%d =%d+%d, %d =%d+%d => %d] _%.0f \n ", blockIdx.x,  warp_id*WARP_SIZE+thread_in_warp, tile_offset_row + split_size_rows*warp_id + k, tile_offset_row,tile_offset_row + split_size_rows*warp_id + k, tile_offset_col + l,tile_offset_col,l, (tile_offset_row + split_size_rows*warp_id + k) * n + tile_offset_col + l,  resArray[k*tile_size_rows + l]);
                     Cd[(tile_offset_row + split_size_rows*warp_id + k) * n + tile_offset_col + l] = resArray[k*tile_size_rows + l];
                }
            }
        }
    }

    // if(warp_id == 0 && thread_in_warp == 0){
    //     *spinlock += 10;
    //     printf("!8!block %d -- finished, spinlock %d!\n", blockIdx.x, *spinlock);
    // }
    return;

}

int main(int argc, const char *argv[]) {

    // //create root Topology and one node
    // Topology* topo = new Topology();
    // Node* n = new Node(topo,1);
    //
    // cout << "-- Parsing gpu-topo benchmark from file " << gpuTopoPath << endl;
    // if(parseGpuTopo((Component*)n, gpuTopoPath, 0, ";") != 0) { //adds topo to a next node
    //     return 1;
    // }
    // cout << "-- End parseGpuTopo" << endl;

    float *A,*B,*C,*D; /* arrays for matrices */
    //int n; /* n=matrix size */
    int n = atoi(argv[1]);
    printf("Matrix mult. of size %d : \n", n);

    high_resolution_clock::time_point t_start, t_end;
    int size = n*n*sizeof(float);
    A = (float *) malloc(size);
    dstMatrix(A,n);
    B = (float *) malloc(size);
    dstMatrix(B,n);
    C = (float *) malloc(size);
    zeroMatrix(C,n);
    D = (float *) malloc(size);
    zeroMatrix(D,n);

    float *Ad, *Bd, *Cd, *Dd;

    hipMalloc((void**)&Ad, size); checkCUDAError("allocate memory for A");
    hipMalloc((void**)&Bd, size); checkCUDAError("allocate memory for B");
    hipMalloc((void**)&Cd, size); checkCUDAError("allocate memory for C");
    hipMalloc((void**)&Dd, size); checkCUDAError("allocate memory for D");

    hipMemcpy(Ad,A, size, hipMemcpyHostToDevice); checkCUDAError("memory of A not transferred");
    hipMemcpy(Bd,B, size, hipMemcpyHostToDevice); checkCUDAError("memory of B not transferred");
    hipMemcpy(Cd,C, size, hipMemcpyHostToDevice); checkCUDAError("memory of C not transferred");
    hipMemcpy(Dd,D, size, hipMemcpyHostToDevice); checkCUDAError("memory of D not transferred");

    //n=4000
    int gpu_num_cores = 3840;
    int gpu_num_sm = 30;
    int oversubscribe = 2;
    int shared_mem_size = 96 * 1024 * 0.9;

    int num_cores_per_sm = gpu_num_cores / gpu_num_sm;
    int num_warps_per_sm = num_cores_per_sm / WARP_SIZE;
    printf("matrix size %d * %d , num warps %d\n", n, n, num_warps_per_sm*gpu_num_sm);

    int num_row_splits = num_warps_per_sm * gpu_num_sm * oversubscribe;
    int split_size_rows = (n +num_row_splits-1)/num_row_splits;
    int split_size_cols = WARP_SIZE;
    printf("split dims %d rows x %d cols \n", split_size_rows, split_size_cols);

    int splits_per_tile_rows = num_warps_per_sm;
    int tile_size_rows = split_size_rows * num_warps_per_sm;

    int splits_per_tile_cols = shared_mem_size / (2 * oversubscribe * tile_size_rows * WARP_SIZE * sizeof(float));
    int tile_size_cols = splits_per_tile_cols * WARP_SIZE;

    int sp = 0;
    int* spinlock_d;
    hipMalloc((void**)&spinlock_d, sizeof(int)); checkCUDAError("allocate memory for sp");
    hipMemcpy(spinlock_d,&sp,  sizeof(int), hipMemcpyHostToDevice); checkCUDAError("memory of sp not transferred");


    printf("num splits per tile = %d row x %d col \n", splits_per_tile_rows, splits_per_tile_cols);
    printf("\ntile: %d rows * %d cols = %d bytes \n",tile_size_rows, tile_size_cols, tile_size_cols * tile_size_rows * 4);
    printf("grid: %d x %d-row split-sets \n", gpu_num_sm * oversubscribe, split_size_rows*num_warps_per_sm);
    int num_tiles_col = (n+tile_size_cols-1) / tile_size_cols;
    int num_tiles_row = (n+tile_size_rows-1) / tile_size_rows;
    dim3 block_dim(WARP_SIZE, num_warps_per_sm);
    dim3 grid_dim(oversubscribe * gpu_num_sm);//gpu_num_sm * oversubscribe
    //dim3 grid_dim(num_tiles_col, num_tiles_row);//gpu_num_sm * oversubscribe
    unsigned block_shared_mem_size = tile_size_rows * tile_size_cols * sizeof(float) * 2;
    t_start = high_resolution_clock::now();
    matrixMultKernel_cache<<<grid_dim,block_dim, block_shared_mem_size>>>(Ad,Bd,Cd,n,tile_size_rows, tile_size_cols, split_size_rows, splits_per_tile_cols, num_tiles_col, spinlock_d);
    hipDeviceSynchronize();
    t_end = high_resolution_clock::now();
    std::cout << "time[us]: " << (t_end.time_since_epoch().count()-t_start.time_since_epoch().count())/1000 << std::endl;

    checkCUDAError("matrixMultKernel_cache failed");
    hipMemcpy(C,Cd, size, hipMemcpyDeviceToHost); checkCUDAError("memory of C not transferred back");

    //printMatrix("C", C, n);
    t_start = high_resolution_clock::now();
    CPU_matrixMult(A,B,D,n,1);
    t_end = high_resolution_clock::now();
    std::cout << "CPU time[us]: " << (t_end.time_since_epoch().count()-t_start.time_since_epoch().count())/1000 << std::endl;

    checkMatrix(C,D,n);
//////////
    //
    // int tile_size_rows = WARP_SIZE;//TODO threads_per_sm?
    // int tile_size_cols = shared_mem_size / (2 * oversubscribe * tile_size_rows * sizeof(float));
    // printf("\ntile dim %d x %d = %d bytes \n",tile_size_rows, tile_size_cols, tile_size_cols * tile_size_rows * 4);
    //
    // dim3 block_dim(tile_size_cols, tile_size_rows);
    // dim3 grid_dim(oversubscribe, gpu_num_sm * num_warps_per_sm);//gpu_num_sm * oversubscribe
    // unsigned shared_mem_size = tile_size_rows * tile_size_cols * sizeof(float) * 2;
    // matrixMultKernel_cache2<<<dimGrid,dimBlock, shared_mem_size >>>(Ad,Bd,Cd,n);
    //

    return 0;


    //
    //
    // int num_cores = 2000;
    // int tile_size_bytes = 24588/64;// = //24588/64 B per core
    //
    // float *A,*B,*C,*D; /* arrays for matrices */
    // //int n; /* n=matrix size */
    // n = atoi(argv[1]);
    // printf("Matrix mult. of size %d : \n", n);
    //
    // int size = n*n*sizeof(float);
    // A = (float *) malloc(size);
    // dstMatrix(A,n);
    // B = (float *) malloc(size);
    // dstMatrix(B,n);
    // C = (float *) malloc(size);
    // zeroMatrix(C,n);
    // D = (float *) malloc(size);
    // zeroMatrix(D,n);
    //
    // float *Ad, *Bd, *Cd, *Dd;
    //
    // cudaMalloc((void**)&Ad, size); checkCUDAError("allocate memory for A");
    // cudaMalloc((void**)&Bd, size); checkCUDAError("allocate memory for B");
    // cudaMalloc((void**)&Cd, size); checkCUDAError("allocate memory for C");
    // cudaMalloc((void**)&Dd, size); checkCUDAError("allocate memory for D");
    //
    // cudaMemcpy(Ad,A, size, cudaMemcpyHostToDevice); checkCUDAError("memory of A not transferred");
    // cudaMemcpy(Bd,B, size, cudaMemcpyHostToDevice); checkCUDAError("memory of B not transferred");
    // cudaMemcpy(Cd,C, size, cudaMemcpyHostToDevice); checkCUDAError("memory of C not transferred");
    // cudaMemcpy(Dd,D, size, cudaMemcpyHostToDevice); checkCUDAError("memory of D not transferred");
    //
    // int num_cores_1dim = sqrt(num_cores);
    // if(num_cores_1dim * num_cores_1dim < num_cores)
    //     num_cores_1dim+=1;
    //
    // int tile_size = n/num_cores_1dim;
    // if(tile_size * num_cores_1dim < n)
    //     tile_size+=1;
    //
    //
    // dim3 dimBlock((num_cores_1dim>32?32:num_cores_1dim), (num_cores_1dim>32?32:num_cores_1dim));
    // dim3 dimGrid( (num_cores_1dim%32==0?num_cores_1dim/32:num_cores_1dim/32+1), (num_cores_1dim%32==0?num_cores_1dim/32:num_cores_1dim/32+1) );
    // printf("launching kernel: matrix %d x %d, each thread computes tile=%d x %d, there is %d cores in each dimension (blockDim %d, gridDim %d) \n", n, n, tile_size, tile_size, num_cores_1dim, (num_cores_1dim>32?32:num_cores_1dim), (num_cores_1dim%32==0?num_cores_1dim/32:num_cores_1dim/32+1) );
    //
    // matrixMultKernel_global<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n, tile_size);
    // cudaMemcpy(C,Cd, size, cudaMemcpyDeviceToHost); checkCUDAError("memory of C not transferred back");
    //
    // printf("matrixMultKernel_global done \n");
    //
    // matrixMultKernel_tiled<<<dimGrid,dimBlock>>>(Ad,Bd,Dd,n, tile_size, 7);
    // cudaMemcpy(D,Dd, size, cudaMemcpyDeviceToHost); checkCUDAError("memory of D not transferred back");
    //
    // //printMatrix("C", C, n);
    // checkMatrix(D,C,n);
    //
    // cudaFree(Ad);
    // cudaFree(Bd);
    // cudaFree(Cd);
    //
    // return(0);
    //


}

__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n, int tile_size)
{
    int first_row_index = threadIdx.x * tile_size * n;
    int first_col_index = threadIdx.y * tile_size;

//    if(threadIdx.x==19)
//        printf("index first_row 7, accessing elems row %d to %d col %d to %d \n", first_row_index/n, first_row_index/n +tile_size, first_col_index, first_col_index + tile_size);

    for(int row=0; row < tile_size && row + threadIdx.x * tile_size < n ; row++)
    {
        for(int col=0; col < tile_size && col + threadIdx.y * tile_size < n; col++)
        {
            for(int j=0; j < n ; j++)
            {
                Cd[first_row_index + row*n + first_col_index + col] += Ad[first_row_index + row*n + j]* Bd[j*n + first_col_index + col];
            }
        }
    }
}


__global__ void matrixMultKernel_optimized(float* Ad, float* Bd, float* Cd, int n, int tile_size_y, int tiles_per_warp)
{
    // cuda_warps = 8*GPU_cores / 32
    // cuda_warps/SM = cores-per-SM/32*8
    // 1 tile - x=to fit into shared memory on SM, y=32
    // num-tiles = n/x * n/y
    // num-tiles-per-warp = num-tiles/cuda_warps

    // int row_in_tile = threadIdx.x;
    //
    // __shared__ float Ads[32][tile_size_y];
    // __shared__ float Bds[32][tile_size_y];
    //
    // float c_part = 0;
    // for(int col = 0; col < tile_size_y; col ++)
    // {
    //     c_part += Ad[ row_in_tile * n + col ] * B[ row_in_tile * n + col ]
    // }
}


__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n, int tile_size, int core_tile_size)
{
    int first_row_index = threadIdx.x * tile_size * n;
    int first_col_index = threadIdx.y * tile_size;

    for(int row=0; row < tile_size && row + threadIdx.x * tile_size < n ; row+=core_tile_size)
    {
        for(int col=0; col < tile_size && col + threadIdx.y * tile_size < n; col+=core_tile_size)
        {
            for(int j=0; j < n ; j+=core_tile_size)
            {
                for(int row_tile=row; row_tile < row+core_tile_size && row_tile + threadIdx.x * tile_size < n ; row_tile++)
                {
                    for(int col_tile=col; col_tile < col+core_tile_size && col_tile + threadIdx.y * tile_size < n; col_tile++)
                    {
                        for(int j_tile=j; j_tile < j+core_tile_size && j_tile < n ; j_tile++)
                        {
                            Cd[first_row_index + row_tile*n + first_col_index + col_tile] += Ad[first_row_index + row_tile*n + j_tile]* Bd[j_tile*n + first_col_index + col_tile];
                        }
                    }
                }
            }
        }
    }
}


/*
 * helper function to check for errors in CUDA calls
 * source: NVIDIA
 */
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();

    if( hipSuccess != err) {
        fprintf(stderr, "\nCuda error (%s): %s.\n", msg, hipGetErrorString( err) );
        //exit(-1);
    }
}

/* print Matrix */
void printMatrix(const char* name, float *A, int n)
{
   int i,k;

   printf("Matrix %s (size %d)\n",name,n);

   for (i=0; i<n; i++) {
     for (k=0; k<n; k++) {
       printf("%f ", A[i*n+k]);
     }

     printf("\n");
   }
}

void checkMatrix(float *to_check, float *reference, int n)
{
    int i,k;
    for (i=0; i<n; i++) {
      for (k=0; k<n; k++) {
        if(to_check[i*n+k] != reference[i*n+k]){
            printf("[%d,%d] %f vs ref %f \n", i,k,to_check[i*n+k],  reference[i*n+k] );
            return;
        }
      }
  }
}

/* set Matrix values to zero */
void zeroMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
	    A[i*n+k] = 0;
}

/* initialise Matrix: discrete Sine Transform */
void dstMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
        A[i*n+k] = (i+1)*10+k+1;
	    //A[i*n+k] = sin( ((i+1)*(k+1)*M_PI)/(n+1));
}

void CPU_matrixMult(float *A, float *B, float *C, int n, int repeats) {
	int i,j,k;
    float tmp;

	for(int r=0; r<repeats; r++) {
    	for (i=0; i<n; i++) {
			for (j=0; j<n; j++) {
				tmp = A[i*n+j];

				for (k=0; k<n; k++) {
					C[i*n+k] += tmp * B[j*n+k];
				}
    		}
		}
    }
}



// __global__ void matrixMultKernel_cache(float* Ad, float* Bd, float* Cd, int n, int tile_size_rows, int tile_size_cols, int split_size_rows, int splits_per_tile_cols, int num_tiles_col, int* spinlock )
// {
//     extern __shared__ float array[];
//     float *Ad_shared = array;
//     float *Bd_shared = &array[tile_size_rows * tile_size_cols];
//
//     int thread_in_warp = threadIdx.x;
//     int warp_id = threadIdx.y;
//
//     float* resArray = (float*)malloc(sizeof(float)*split_size_rows * tile_size_rows);
//     // float* temp_res;
//     // if(warp_id == 0 && thread_in_warp == 0)
//     // {
//     //     temp_res= (float*)malloc(sizeof(float)*tile_size_rows * tile_size_rows);
//     // }
//
//     if(blockIdx.x * tile_size_rows >= n){
//          // if(warp_id == 0 && thread_in_warp == 0)
//          //     printf("!!block %d skipping -- indexes %d and on\n", blockIdx.x, blockIdx.x * tile_size_rows);
//         return;
//     }
//
// while (* spinlock < blockIdx.x*10) {
//
// }
//     for(int tile_offset_col = 0 ; tile_offset_col < n; tile_offset_col+=tile_size_rows)
//     {//one loop of this fills tile_size_rows*tile_size_rows matrix
//
//         //int tile_col=tile_col_num;//J
//         int tile_offset_row=blockIdx.x*tile_size_rows;//I
//
//         // if(warp_id == 0 && thread_in_warp == 0)
//         //     printf("1 block %d --(%d) processing tile col indexes %d to %d\n", blockIdx.x, tile_size_rows * tile_size_cols, tile_offset_col, tile_offset_col+tile_size_rows -1);
//
//         for(int i = 0; i < split_size_rows * tile_size_rows; i++ )
//         {
//             resArray[i] = 0;
//         }
//         // if(warp_id == 0 && thread_in_warp == 0)
//         // {
//         //     for(int i = 0; i < tile_size_rows * tile_size_rows; i++ )
//         //     {
//         //         temp_res[i] = 0;
//         //     }
//         // }
//         // if(warp_id == 0 && thread_in_warp == 0)
//         // {
//         //     printf("block %d -- setting temp_res to 0 (%d elems)\n", blockIdx.x, tile_size_rows * tile_size_rows);
//         //     for(int l = 0; l<tile_size_rows; l++)
//         //     {
//         //         for(int m = 0; m<tile_size_rows; m++)
//         //         {
//         //              printf("x%.0f   ",temp_res[m*tile_size_rows + l]);
//         //         }
//         //         printf("\n");
//         //     }
//         // }
//
//
//         int a_num_rows = ((tile_offset_row+tile_size_rows <= n) ? tile_size_rows : n - tile_offset_row);
//         int b_num_cols = ((tile_offset_col + tile_size_rows <= n) ? tile_size_rows : n - tile_offset_col);
//
//         for (int t=0; t<num_tiles_col; t++)
//         {
//             int a_num_cols_b_num_rows = ((tile_size_cols*(t+1) <= n) ? tile_size_cols : n - tile_size_cols*t );
//
//             // if(warp_id == 0 && thread_in_warp == 0)
//             //     printf("2 block %d tile col %d, t %d -- a=%dx%d , b=%dx%d -- A[%d-%d, %d-%d], B[%d-%d, %d-%d]\n", blockIdx.x, tile_offset_col, t, a_num_rows, a_num_cols_b_num_rows, a_num_cols_b_num_rows, b_num_cols,      tile_offset_row, tile_offset_row+a_num_rows, t*tile_size_cols, t*tile_size_cols+a_num_cols_b_num_rows,        t*tile_size_cols, t*tile_size_cols + a_num_cols_b_num_rows,tile_offset_col,tile_offset_col+b_num_cols);
//
//             for(int split_col = 0; split_col < splits_per_tile_cols ; split_col++)
//             {
//                 int elem_col = split_col*WARP_SIZE + thread_in_warp;
//                 if(elem_col >= a_num_cols_b_num_rows) break;
//                 for(int row = 0; row < split_size_rows; row++ )
//                 {
//                     int elem_row = warp_id * split_size_rows + row;
//                     if(elem_row < a_num_rows)
//                         Ad_shared[elem_row*tile_size_cols + elem_col] = Ad[(tile_offset_row + elem_row)*n + t*tile_size_cols + elem_col];
//                     if(elem_row < b_num_cols)
//                         Bd_shared[elem_row*tile_size_cols + elem_col] = Bd[elem_row + tile_offset_col + (t*tile_size_cols + elem_col)*n];
//                 }
//             }
//             __syncthreads();
//
//
//
//             // if(warp_id == 0 && thread_in_warp == 0)
//             //     printf("3 block %d tile col %d -- filled shared array t %d\n", blockIdx.x, tile_offset_col, t);
//
//             // if(warp_id == 0 && thread_in_warp == 0)
//             // {
//             //     for(int row = 0; row < tile_size_rows; row++ )
//             //     {
//             //         for(int col = 0; col < 12 ; col++)
//             //         {
//             //             printf("%.0f _ %.0f  ", Ad_shared[row*tile_size_cols + col], Bd_shared[row*tile_size_cols + col]);
//             //         }
//             //         printf("\n");
//             //     }
//             // }
//
//             for(int k=0; k < a_num_rows - split_size_rows*warp_id && k< split_size_rows  ; k++)
//             {
//                 for(int l = 0; l<b_num_cols ; l++)
//                 {
//                     for(int m = thread_in_warp; m<a_num_cols_b_num_rows; m+=WARP_SIZE)
//                     {
//                         //printf("block %d thread %d tile col %d --temp_res[%d][%d][%d](%f)+=As[%d][%d][%d](%f) * Bs[%d][%d][%d](%f) \n", blockIdx.x,  warp_id*WARP_SIZE+thread_in_warp, tile_offset_col, k,l,k*tile_size_rows + l,resArray[k*tile_size_rows + l], split_size_rows*warp_id + k,m,(split_size_rows*warp_id + k)*tile_size_cols + m,Ad_shared[(split_size_rows*warp_id + k)*tile_size_cols + m], l,m, l * tile_size_cols + m, Bd_shared[l * tile_size_cols + m]);
//                         resArray[k*tile_size_rows + l] += Ad_shared[(split_size_rows*warp_id + k) * tile_size_cols + m] * Bd_shared[l * tile_size_cols + m];
//                         //resArray[k*tile_size_rows + l] += Ad_shared[k*tile_size_cols + m] * Bd_shared[l * tile_size_cols + m];
//                     }
//                 }
//             }
//
//             // if(warp_id == 0 && thread_in_warp == 0)
//             // {
//             //     for(int k=0; k<a_num_rows  ; k++)
//             //     {
//             //         // if(t*tile_size_cols + k >= n)
//             //         // {
//             //         //     if(warp_id == 0 && thread_in_warp == 0)
//             //         //          printf("4 block %d tile col %d t %d --k=%d -- break! \n", blockIdx.x, tile_offset_col, t, k);
//             //         //     break;
//             //         // }
//             //
//             //         for(int l = 0; l<b_num_cols ; l++)
//             //         {
//             //             for(int m = 0; m<a_num_cols_b_num_rows; m++)
//             //             {
//             //                 //printf("block %d tile col %d --temp_res[%d][%d][%d](%f)+=As[%d][%d][%d](%f) * Bs[%d][%d][%d](%f) \n", blockIdx.x, tile_offset_col, k,l,k*tile_size_rows + l,temp_res[k*tile_size_rows + l], k,m,k*tile_size_cols + m,Ad_shared[k*tile_size_cols + m], l,m, l * tile_size_cols + m, Bd_shared[l * tile_size_cols + m]);
//             //                 temp_res[k*tile_size_rows + l] += Ad_shared[k*tile_size_cols + m] * Bd_shared[l * tile_size_cols + m];
//             //             }
//             //         }
//             //     }
//             //     // for(int k=0; k<tile_size_rows && k<a_num_rows  ; k++)
//             //     // {
//             //     //     if(t*tile_size_cols + k >= n)
//             //     //     {
//             //     //         if(warp_id == 0 && thread_in_warp == 0)
//             //     //              printf("4 block %d tile col %d t %d --k=%d -- break! \n", blockIdx.x, tile_offset_col, t, k);
//             //     //         break;
//             //     //     }
//             //     //
//             //     //     for(int l = 0; l<tile_size_rows && l<b_num_cols ; l++)
//             //     //     {
//             //     //         for(int m = 0; m<tile_size_cols && m<a_num_cols_b_num_rows; m++)
//             //     //         {
//             //     //             //printf("block %d tile col %d --temp_res[%d][%d][%d](%f)+=As[%d][%d][%d](%f) * Bs[%d][%d][%d](%f) \n", blockIdx.x, tile_offset_col, k,l,k*tile_size_rows + l,temp_res[k*tile_size_rows + l], k,m,k*tile_size_cols + m,Ad_shared[k*tile_size_cols + m], l,m, l * tile_size_cols + m, Bd_shared[l * tile_size_cols + m]);
//             //     //             temp_res[k*tile_size_rows + l] += Ad_shared[k*tile_size_cols + m] * Bd_shared[l * tile_size_cols + m];
//             //     //         }
//             //     //     }
//             //     // }
//             // }
//             // if(warp_id == 0 && thread_in_warp == 0)
//             //     printf("5 block %d tile col %d -- finished computation number t %d\n", blockIdx.x, tile_offset_col, t);
//             __syncthreads();
//
//         }
//
//         //fan-in from all threads in one warp that have co-computed the same indexes (one row)
//         for(int row = 0; row < tile_size_rows; row++ )
//         {
//             for(int col = 0; col < split_size_rows ; col++)
//             {
//                 if(resArray[row*split_size_rows + col] != 0  && thread_in_warp==0)
//                     printf("--thread %d [%d,%d] _%.0f \n", warp_id*WARP_SIZE+thread_in_warp, row, col, resArray[row*split_size_rows + col]);
//             }
//         }
//         for(int i = 0; i < split_size_rows * tile_size_rows; i++ )
//         {
//             for (int offset = 16; offset > 0; offset /= 2)
//             {
//                 resArray[i] += __shfl_down_sync(FULL_MASK, resArray[i], offset,32);
//             }
//         }
//         __syncthreads();
//         for(int row = 0; row < tile_size_rows; row++ )
//         {
//             for(int col = 0; col < split_size_rows ; col++)
//             {
//                 if(resArray[row*split_size_rows + col] != 0 && thread_in_warp==0)
//                     printf("thread %d [%d,%d] _%.0f \n", warp_id*WARP_SIZE+thread_in_warp, row, col, resArray[row*split_size_rows + col]);
//             }
//         }
//
//         if(thread_in_warp == 0)
//         {
//             for(int k=0; k < a_num_rows - split_size_rows*warp_id && k< split_size_rows  ; k++)
//             {
//                 for(int l = 0; l<b_num_cols ; l++)
//                 {
//                      printf("6 -----===== block %d thread %d filling index [%d =%d+%d, %d =%d+%d => %d] _%.0f \n ", blockIdx.x,  warp_id*WARP_SIZE+thread_in_warp, tile_offset_row + split_size_rows*warp_id + k, tile_offset_row,tile_offset_row + split_size_rows*warp_id + k, tile_offset_col + l,tile_offset_col,l, (tile_offset_row + split_size_rows*warp_id + k) * n + tile_offset_col + l,  resArray[k*tile_size_rows + l]);
//                      Cd[(tile_offset_row + split_size_rows*warp_id + k) * n + tile_offset_col + l] = resArray[k*tile_size_rows + l];
//                 }
//             }
//         }
//         // if(thread_in_warp == 0)
//         // {
//         //     for(int k = tile_size_rows*warp_id ; k<a_num_rows && k<tile_size_rows*(warp_id+1) ; k++)
//         //     {
//         //         for(int l = 0; l<b_num_cols; l++)
//         //         {
//         //             printf("6 -----===== block %d thread %d filling index [%d =%d+%d, %d =%d+%d => %d] _%.0f \n ", blockIdx.x,  warp_id*WARP_SIZE+thread_in_warp, tile_offset_row+k, tile_offset_row,k, tile_offset_col + l,tile_offset_col,l, (tile_offset_row +k) * n + tile_offset_col + l,  resArray[k*tile_size_rows + l]);
//         //              Cd[(tile_offset_row +k) * n + tile_offset_col + l] = resArray[k*tile_size_rows + l];
//         //         }
//         //     }
//         // }
//
//
//         // if(warp_id == 0 && thread_in_warp == 0)
//         // {
//         //     for(int row = 0; row < tile_size_rows; row++ )
//         //     {
//         //         for(int col = 0; col < tile_size_rows ; col++)
//         //         {
//         //             printf("_%.0f ", temp_res[row*tile_size_rows + col]);
//         //         }
//         //         printf("\n");
//         //     }
//         // }
//     //x    // if(warp_id == 0 && thread_in_warp == 0)
//         // {
//         //     for(int k = 0; k<a_num_rows; k++)
//         //     {
//         //         for(int l = 0; l<b_num_cols; l++)
//         //         {
//         //             //printf("6 -----===== block %d filling index [%d =%d+%d, %d =%d+%d => %d] _%.0f \n ", blockIdx.x,  tile_offset_row+k, tile_offset_row,k, tile_offset_col + l,tile_offset_col,l, (tile_offset_row +k) * n + tile_offset_col + l,  temp_res[k*tile_size_rows + l]);
//         //              Cd[(tile_offset_row +k) * n + tile_offset_col + l] = temp_res[k*tile_size_rows + l];
//         //         }
//         //     }
//         // }
//
//         // if(warp_id == 0 && thread_in_warp == 0)
//         //     printf("7 block %d tile col %d -- finished!\n", blockIdx.x, tile_offset_col);
//     }
//
//     if(warp_id == 0 && thread_in_warp == 0){
//         *spinlock += 10;
//         printf("!8!block %d -- finished, spinlock %d!\n", blockIdx.x, *spinlock);
//     }
//     return;
//
// }
